#include "hip/hip_runtime.h"
﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <inttypes.h>
#include <stdio.h>
#include <stdlib.h>

#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"

#define SUPERBLOCK_SIZE 4096 //32K/8 bits = 4 K
#define FCB_SIZE 32 //32 bytes per FCB
#define FCB_ENTRIES 1024
#define VOLUME_SIZE 1085440 //4096+32768+1048576
#define STORAGE_BLOCK_SIZE 32

#define MAX_FILENAME_SIZE 20
#define MAX_FILE_NUM 1024
#define MAX_FILE_SIZE 1048576

#define FILE_BASE_ADDRESS 36864 //4096+32768


// data input and output
__device__ __managed__ uchar input[MAX_FILE_SIZE];
__device__ __managed__ uchar output[MAX_FILE_SIZE];

// volume (disk storage)
__device__ __managed__ uchar volume[VOLUME_SIZE];



__device__ void user_program(FileSystem *fs, uchar *input, uchar *output);

__global__ void mykernel(uchar *input, uchar *output) {

  // Initilize the file system	
  FileSystem fs;
  fs_init(&fs, volume, SUPERBLOCK_SIZE, FCB_SIZE, FCB_ENTRIES, 
			VOLUME_SIZE,STORAGE_BLOCK_SIZE, MAX_FILENAME_SIZE, 
			MAX_FILE_NUM, MAX_FILE_SIZE, FILE_BASE_ADDRESS);

  // user program the access pattern for testing file operations
  user_program(&fs, input, output);
}

__host__ void write_binaryFile(char *fileName, void *buffer, int bufferSize)
{
	FILE *fp;
	fp = fopen(fileName, "wb");
	fwrite(buffer, 1, bufferSize, fp);
	fclose(fp);
}

__host__ int load_binaryFile(char *fileName, void *buffer, int bufferSize)
{
	FILE *fp;
	fp = fopen(fileName, "rb");

	if (!fp)
	{
		printf("***Unable to open file %s***\n", fileName);
		exit(1);
	}

	//Get file length
	fseek(fp, 0, SEEK_END);
	int fileLen = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	if (fileLen > bufferSize)
	{
		printf("****invalid testcase!!****\n");
		printf("****software warrning: the file: %s size****\n", fileName);
		printf("****is greater than buffer size****\n");
		exit(1);
	}

	//Read file contents into buffer
	fread(buffer, fileLen, 1, fp);
	fclose(fp);
	return fileLen;
}

int main() {
  hipError_t cudaStatus;
  load_binaryFile(DATAFILE, input, MAX_FILE_SIZE);

  // Launch to GPU kernel with single thread
  mykernel<<<1, 1>>>(input, output);

  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "mykernel launch failed: %s\n",
            hipGetErrorString(cudaStatus));
    return 0;
  }

  hipDeviceSynchronize();
  hipDeviceReset();

  write_binaryFile(OUTFILE, output, MAX_FILE_SIZE);


  return 0;
}
