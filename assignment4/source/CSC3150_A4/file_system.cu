﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;
__device__ __managed__ u32 global_storage_end = 0;

__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

  //init superblock
  for (int i = 0; i < SUPERBLOCK_SIZE; i++) {
	  fs->volume[i] = 0;
  }

  //init FCB
  //FCB: 0-19   name
  //	 20-21  address 
  //	 22-23  size (valid bit at 22)
  //	 24-25  created time
  //	 26-27  modified time
  for (int i = 0; i < FCB_ENTRIES; i++) {
	  fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 22] = 0xff; //set to invalid
  }
}

__device__ bool compare_string(char * s1, char * s2) {
	while (*s1 != '\0' && *s2 != '\0' && *s1 == *s2) {
		s1++;
		s2++;
	}

	if (*s1 == '\0' && *s2 == '\0') {
		return true;
	}
	return false;
}

__device__ void prt_string(uchar * s) {
	while (*s != '\0') {
		printf("%c", (char) *s);
		s++;
	}
}

__device__ u32 set_superblock(FileSystem *fs, int block_addr, int type) {
	if (block_addr < 0 || block_addr >= 1024) {
		return 0xffffffff;
	}
	int row = block_addr / 8;
	int column = block_addr % 8;
	uchar mask;
	mask = (1 << column);

	if (type == 0) {
		mask = ~mask;
		fs->volume[row] &= mask;
	}
	else {
		fs->volume[row] |= mask;
	}
	return 0;
}

__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	u32 FCB_pt = 0x10000000;
	int empty_entry = -1;
	for (int i = 0; i < fs->FCB_ENTRIES; i++) {
		//checking valid files
		if (fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 22] != 0xff) {
			if (compare_string((char *) &fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i], s)) {
				FCB_pt = i;
				break;
			}
		}
		else {
			if (empty_entry == -1) empty_entry = i;
		}
	}

	//find the file
	if (FCB_pt != 0x10000000) {
		return FCB_pt;
	}
	else {
		if (op == G_WRITE) {
			if (empty_entry == -1) {
				printf("files number reach the maximun\n");
			}
			else {
				uchar temp = *s;
				char * ss = s;
				int FCB_entry_address = fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * empty_entry;

				//set new file name
				int name_count = 0;
				while (temp != '\0') {
					fs->volume[FCB_entry_address + name_count] = temp;
					ss++;
					name_count++;
					temp = *ss;
					if (name_count == fs->MAX_FILENAME_SIZE) {
						printf("file name exceed the limit\n");
						return FCB_pt;
					}
				}
				
				fs->volume[FCB_entry_address + name_count] = '\0';

				//set size
				fs->volume[FCB_entry_address + 22] = 0;
				fs->volume[FCB_entry_address + 23] = 0;

				//set create time
				fs->volume[FCB_entry_address + 24] = gtime / 256;
				fs->volume[FCB_entry_address + 25] = gtime % 256;

				//set modified time
				fs->volume[FCB_entry_address + 26] = gtime / 256;
				fs->volume[FCB_entry_address + 27] = gtime % 256;
				gtime++;

				FCB_pt = empty_entry;
			}
		}
		return FCB_pt;
	}
}

__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	//check whether fp is valid
	if (fp == 0x10000000 || fp >= 1024) {
		printf("invalid file pointer\n");
		return;
	}

	if (fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 22] == 0xff) {
		printf("invalid file\n");
		return;
	}

	//check whether size exceed the file size
	int file_size = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 22];
	file_size *= 256;
	file_size += fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 23];
	if (size > file_size) {
		printf("read exceed the file size\n");
		return;
	}

	int storage_addr = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 20];
	storage_addr *= 256;
	storage_addr += fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 21];
	storage_addr *= fs->STORAGE_BLOCK_SIZE;
	storage_addr += fs->FILE_BASE_ADDRESS;

	for (int i = 0; i < size; i++) {
		output[i] = fs->volume[storage_addr + i];
	}
}


__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	if (size >= fs->MAX_FILE_SIZE) {
		printf("new size exceed the limit of the file size\n");
		return 0x10000000;
	}

	//check whether fp is valid
	if (fp == 0x10000000 || fp >= 1024) {
		printf("invalid file pointer\n");
		return 0x10000000;
	}

	if (fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 22] == 0xff) {
		printf("invalid file\n");
		return 0x10000000;
	}

	int new_addr = fs_mount(fs, size, fp);

	for (int i = 0; i < size; i++) {
		fs->volume[new_addr + i] = input[i];
	}

	int shift_block = size / fs->STORAGE_BLOCK_SIZE;
	if (size % fs->STORAGE_BLOCK_SIZE != 0) {
		shift_block++;
	}

	int old_size = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 22] * 256;
	old_size += fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 23];

	global_storage_end += shift_block * fs->STORAGE_BLOCK_SIZE;

	int start_block = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 20];
	start_block = start_block * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 21];

	//set superblock
	for (int i = 0; i < shift_block; i++) {
		set_superblock(fs, start_block + i, 1);
	}

	start_block = global_storage_end / fs->STORAGE_BLOCK_SIZE;

	//clear superblock
	if (size < old_size) {
		int remain_blocks = old_size - size;
		remain_blocks = remain_blocks / fs->STORAGE_BLOCK_SIZE;
		if (remain_blocks % fs->STORAGE_BLOCK_SIZE != 0) {
			remain_blocks++;
		}

		for (int i = 0; i < remain_blocks; i++) {
			set_superblock(fs, start_block + i, 0);
		}
	}
	
	int fp_modified_time = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 26];
	fp_modified_time = fp_modified_time * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 27];
	for (int i = 0; i < fs->FCB_ENTRIES; i++) {
		if (i != fp && fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 22] != 0xff) {
			int original_modifid_time = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 26];
			original_modifid_time = original_modifid_time * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 27];

			if (original_modifid_time > fp_modified_time) {
				original_modifid_time--;
				fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 26] = original_modifid_time / 256;
				fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 27] = original_modifid_time % 256;
			}
		}
	}

	fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 22] = size / 256;
	fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 23] = size % 256;
	

	fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 26] = (gtime - 1) / 256;
	fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 27] = (gtime - 1) % 256;
	
	return 0;
}

__device__ bool less_than(FileSystem *fs, u32 fp1, u32 fp2) {
	int size1 = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp1 + 22];
	size1 = size1 * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp1 + 23];

	int size2 = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp2 + 22];
	size2 = size2 * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp2 + 23];

	if (size1 < size2) {
		return true;
	}
	else if (size1 > size2) {
		return false;
	}
	else {
		int time1 = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp1 + 24];
		time1 = time1 * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp1 + 25];

		int time2 = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp2 + 24];
		time2 = time2 * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp2 + 25];

		if (time1 > time2) {
			return true;
		}
		return false;
	}
}

__device__ void fs_gsys(FileSystem *fs, int op)
{

	if (op == LS_D) {
		printf("===sort by modified time===\n");
		int cur_file_count = gtime-1;
		for (int i = 0; i < gtime; i++) {
			for (int i = 0; i < fs->FCB_ENTRIES; i++) {
				if (fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 22] != 0xff) {
					int original_modifid_time = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 26];
					original_modifid_time = original_modifid_time * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 27];
					if (original_modifid_time == cur_file_count) {
						uchar* name = &fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i];
						prt_string(name);
						printf("\n");
						cur_file_count--;
						break;
					}
				}
			}
		}
	}
	else {
		printf("===sort by file size===\n");

		int last_max = -1;
		for (int i = 0; i < fs->FCB_ENTRIES; i++) {
			if (fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 22] != 0xff) {
				if (last_max == -1) {
					last_max = i;
				}
				else {
					if (!less_than(fs, i, last_max)) {
						last_max = i;
					}
				}
			}
		}

		uchar* name = &fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * last_max];
		int size = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * last_max + 22];
		size = size * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * last_max + 23];

		prt_string(name);
		printf(" %d\n", size);

		for (int i = 0; i < gtime - 1; i++) {
			int cur_max = -1;
			for (int j = 0; j < fs->FCB_ENTRIES; j++) {
				if (fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 22] != 0xff) {
					if (cur_max == -1 && less_than(fs, j, last_max)) {
						cur_max = j;
					}
					else {
						if (!less_than(fs, j, cur_max) && less_than(fs, j, last_max)) {
							cur_max = j;
						}
					}
				}
			}

			last_max = cur_max;
			uchar* name = &fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * last_max];
			int size = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * last_max + 22];
			size = size * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * last_max + 23];

			prt_string(name);
			printf(" %d\n", size);
		}
	}
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	if (op == RM) {
		int posi = -1;
		for (int i = 0; i < fs->FCB_ENTRIES; i++) {
			uchar* name = &fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i];
			if (compare_string((char *) name, s)) {
				posi = i;
				break;
			}
		}

		if (posi != -1) {
			int shift_size = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * posi + 22];
			shift_size = shift_size * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * posi + 23];
			int shift_block_size = shift_size / fs->STORAGE_BLOCK_SIZE;
			if (shift_size % fs->STORAGE_BLOCK_SIZE != 0) {
				shift_block_size++;
			}

			
			fs_mount(fs, 0, posi);
			u32 block_addr = global_storage_end / fs->STORAGE_BLOCK_SIZE;

			//clear superblock
			for (int i = 0; i < shift_block_size; i++) {
				set_superblock(fs, block_addr + i, 0);
			}

			fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * posi + 22] = 0xff;
			gtime--;

			int rm_time = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * posi + 24];
			rm_time = rm_time * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * posi + 25];

			int rm_time_modified = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * posi + 26];
			rm_time_modified = rm_time_modified * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * posi + 27];

			int file_cnt = 0;
			
			for (int i = 0; i < fs->FCB_ENTRIES; i++) {
				if (fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 22] != 0xff) {
					int rm_time_i = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 24];
					rm_time_i = rm_time_i * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 25];

					int rm_time_modified_i = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 26];
					rm_time_modified_i = rm_time_modified_i * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 27];

					if (rm_time_i > rm_time) {
						rm_time_i--;
						fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 24] = rm_time_i / 256;
						fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 25] = rm_time_i % 256;
					}
					if (rm_time_modified_i > rm_time_modified) {
						rm_time_modified_i--;
						fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 26] = rm_time_modified_i / 256;
						fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 27] = rm_time_modified_i % 256;
					}
					
					file_cnt++;
				}
				if (file_cnt == gtime) break;
			}
		}
	}
}

__device__ u32 fs_mount(FileSystem *fs, int new_size, u32 fp) {

	int old_size = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 22];
	old_size *= 256;
	old_size += fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 23];

	if (old_size == 0) {
		int storage_block_size = global_storage_end / fs->STORAGE_BLOCK_SIZE;

		fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 20] = storage_block_size / 256;
		fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 21] = storage_block_size % 256;
	
		return global_storage_end + fs->FILE_BASE_ADDRESS;
	}

	int storage_addr = 0;
	storage_addr = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 20];
	storage_addr *= 256;
	storage_addr += fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 21];
	storage_addr *= fs->STORAGE_BLOCK_SIZE;
	storage_addr += fs->FILE_BASE_ADDRESS;

	if (old_size == new_size) {
		return storage_addr;
	}

	int shift_size = old_size;
	if (new_size < old_size) {
		shift_size = old_size - new_size;
	}

	shift_size /= fs->STORAGE_BLOCK_SIZE;
	if (old_size % fs->STORAGE_BLOCK_SIZE != 0) {
		shift_size += 1;
	}

	for (int i = 0; i < fs->FCB_ENTRIES; i++) {
		if (i != fp && fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 22] != 0xff) {
			int tmp = fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 20];
			tmp = tmp * 256 + fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 21];
			int tmp_full = tmp * fs->STORAGE_BLOCK_SIZE + fs->FILE_BASE_ADDRESS;

			//change the addressed of the entries behind the entry
			if (tmp_full > storage_addr) {
				//set superblock bits
				tmp -= shift_size;

				for (int i = 0; i < shift_size; i++) {
					set_superblock(fs, tmp + i, 1);
				}

				fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 20] = tmp / 256;
				fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 21] = tmp % 256;
			}
		}
	}

	int move_start_addr = storage_addr;
	if (new_size < old_size) {
		move_start_addr = storage_addr + new_size;
	}

	//shift storage and superblock after the entry together
	for (int i = 0; i < shift_size; i++) {
		int tmp_addr = move_start_addr + i * fs->STORAGE_BLOCK_SIZE;
		for (int j = 0; j < fs->STORAGE_BLOCK_SIZE; j++) {
			fs->volume[tmp_addr + j] = fs->volume[tmp_addr + j + shift_size * fs->STORAGE_BLOCK_SIZE];
		}
	}

	global_storage_end -= shift_size * fs->STORAGE_BLOCK_SIZE;

	if (new_size < old_size) {
		return storage_addr;
	}

	int new_block_addr = global_storage_end / fs->STORAGE_BLOCK_SIZE;

	fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 20] = new_block_addr / 256;
	fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * fp + 21] = new_block_addr % 256;

	return global_storage_end + fs->FILE_BASE_ADDRESS;
}